#include "hip/hip_runtime.h"
#include <stdio.h>
#include "verfiy.h"
#include "conv2d.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

int main(int argc, char**argv)
{
    //从命令行读入参数
    int n = atoi(argv[1]);
    int c = atoi(argv[2]);
    int h = atoi(argv[3]);
    int w = atoi(argv[4]);
    int k = atoi(argv[5]);
    int r = atoi(argv[6]);
    int s = atoi(argv[7]);
    int u = atoi(argv[8]);
    int v = atoi(argv[9]);
    int p = atoi(argv[10]);
    int q = atoi(argv[11]);

    int outh = (h - r + 2*p)/u + 1;
    int outw = (w - s + 2*q)/v + 1;

    half *pIn       = (half*)malloc(n*c*h*w*sizeof(half));           //原始数据
    half *pWeight   = (half*)malloc(k*c*r*s*sizeof(half));           //卷积核
    half *pOut      = (half*)malloc(n*k*outh*outw*sizeof(half));     //存储正确的计算结果
    half *pOut_host = (half*)malloc(n*k*outh*outw*sizeof(half));     //存储你的的计算结果

    half *pIn_device,*pWeight_device,*pOut_device,*pIn_ori,*pWeight_ori,*pOut_ori;
    hipMalloc((void**)&pIn_device, n*c*h*w*sizeof(half));
    hipMalloc((void**)&pWeight_device, k*c*r*s*sizeof(half));
    hipMalloc((void**)&pOut_device, n*k*outh*outw*sizeof(half));
    hipMalloc((void**)&pIn_ori, n*c*h*w*sizeof(half));
    hipMalloc((void**)&pWeight_ori, k*c*r*s*sizeof(half));
    hipMalloc((void**)&pOut_ori, n*k*outh*outw*sizeof(half));
    
    for(int i = 0; i < n*c*h*w; i++)
    {
        pIn[i] = (rand()%255)/255.0;
    }
    
    for(int i = 0; i < k*c*r*s; i++)
    {
        pWeight[i] = (rand()%255)/255.0;
    }
    
    for(int i = 0; i < n*k*outh*outw; i++)
    {
        pOut[i] = 0.0;
        pOut_host[i] = 0.0;
    }
              
    hipMemcpy(pIn_device, pIn, n*c*h*w*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(pWeight_device, pWeight, k*c*r*s*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(pOut_device, pOut, n*k*outh*outw*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(pIn_ori, pIn, n*c*h*w*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(pWeight_ori, pWeight, k*c*r*s*sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(pOut_ori, pOut, n*k*outh*outw*sizeof(half), hipMemcpyHostToDevice);

    /********************step 1*****************************/
    problem_t problem;
    int paramSize;
    kernelInfo_t kernelInfo;

    problem.in        = pIn_device;        
    problem.weight    = pWeight_device;
    problem.out       = pOut_device;             
    problem.n         = n;                             
    problem.c         = c;                             
    problem.h         = h;                             
    problem.w         = w;                             
    problem.k         = k;                             
    problem.r         = r;                             
    problem.s         = s;                             
    problem.u         = u;                             
    problem.v         = v;                             
    problem.p         = p;                             
    problem.q         = q;                               

    /********************************** step 2****************************/
    getParamsize(&problem, &paramSize);
    printf("paramsize:%d\n", paramSize);
    void* param = malloc(paramSize);
    
    getkernelInfo(&problem, &kernelInfo, param);

    dim3 groups(kernelInfo.blockx, kernelInfo.blocky, kernelInfo.blockz);
    dim3 threads(kernelInfo.threadx, kernelInfo.thready, kernelInfo.threadz);
    int ldsSize = kernelInfo.dynmicLdsSize;
        
    /*******************************warm up and get result************************************/
    hipLaunchKernel(reinterpret_cast<const void*>(kernelInfo.kernelPtr),groups,threads,(void**)&param,ldsSize);

    hipMemcpy(pOut_host, pOut_device,  n*k*outh*outw*sizeof(half), hipMemcpyDeviceToHost); 

    /*******************************cost time test************************************/
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    float time_elapsed=0.0;
    
    int iternum = 100;
    for(int i=0; i<iternum; i++)
    {
        hipLaunchKernel(reinterpret_cast<const void*>(kernelInfo.kernelPtr),groups,threads,(void**)&param,ldsSize); 
    }
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);

    printf("time: %f us\n", time_elapsed*1000/iternum);
    hipEventDestroy(start);
    hipEventDestroy(stop);  

    /*******************************verify************************************/
    printf("start verfiy\n");

    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks((outh * outw + 15) / 16, (k + 15) / 16, n);
    Conv2dGpu<<<numBlocks, threadsPerBlock>>>(pIn_ori, pWeight_ori, pOut_ori, n, c, h, w, k, r, s, u, v, p, q);
    hipMemcpy(pOut, pOut_ori,  n*k*outh*outw*sizeof(half), hipMemcpyDeviceToHost); 

    int error=0;
    for(int i=0;i<n*k*outh*outw;i++)
    {
        float your_result = __half2float(pOut_host[i]);
        float right_result = __half2float(pOut[i]);
        if(isnan(right_result)|isinf(right_result)) 
        {
            printf("right result is nan or inf! It is impossible!");
            break;
        }

        if(isnan(your_result)|isinf(your_result)) 
        {
            printf("your result is nan or inf!");
            break;
        }
        if((fabs(your_result - right_result))/ your_result > 0.01)
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, (float)pOut_host[i], (float)pOut[i]);
            error++;
            break;
        }        
    }

    printf("finish,error:%d\n",error);

    hipFree(pIn_device);
    hipFree(pWeight_device);
    hipFree(pOut_device);
    free(param);
    free(pIn);
    free(pWeight);
    free(pOut);
    free(pOut_host);

    return 0;
}